#include "hip/hip_runtime.h"
#include <stdio.h>
// #include <math.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <stdbool.h>
#include "common.h"

// #define L 5  // Lattice size
// #define N (int)pow(L,d) // Number of lattice points

#define dmax 5  // Dimension

#define TYPE double
#define FUNCTION(NAME) NAME ## _d
#include "laplace-x.h"
#undef TYPE
// #define TYPE float
// #include "laplace-x.h"

#include "conjugate-gradient_cpu.h"
double f(int x)
{
    int L = 5;
    return sin(3.14*x/L);
}

int main()
{
    run_test_gc_cpu();
    int L = 5;
    int d = 3;
    int N = (int)pow(L,d);
    double* b = cuda_allocate_field_d(N);
    double* x = cuda_allocate_field_d(N);
    double* x_cpu = (double*)malloc(N*sizeof(double));
    apply_function_gpu_d<<<1000,64>>>(x,f,N,L,d);    
    hipMemcpy(x_cpu,x,N*sizeof(double),hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%f\n",x_cpu[i]);
    }
}
