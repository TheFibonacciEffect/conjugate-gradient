#include "hip/hip_runtime.h"
#include "common.h"
#include <cassert>
#include <math.h>
#include <stdio.h>
#include <string.h>
#define NTHREADS 32
#define TYPE float
// #include <stdlib.h>
// #include <float.h>

// copied to main.cu
__global__ void sum(int *g_idata, int *g_odata) {
  __shared__ int sdata[NTHREADS]; // why do I not need to reserve this?

  // each itteration loads one element from global to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();

  // do reduction in shared memory
  // why cant I do it in the normal memory (whatever that is)?
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    if (tid % (2 * s) == 0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0)
    g_odata[blockIdx.x] = sdata[0];
}

// Copied to main.cu
__global__ void inner_product(TYPE *result, TYPE *a, TYPE *b, int n,
                              int arretmetic) {
  assert(blockDim.x * gridDim.x > n);
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + arretmetic * threadIdx.x;

  if (i > n)
    return;
  for (int j = 0; j < arretmetic && i + j < n; j++) {
    result[tid] += a[i + j] * b[i + j];
  }

  return;
}

__global__ void fill(int *data, int value, const int N) {
  int idx;
  idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (; (idx < N); idx += blockDim.x * gridDim.x)
    data[idx] = value;
}

int main() {
  int N = 10000;
  int nblocks = 32;
  int *A, *zwischenergebnisse, *ergebnis, *result_cpu;
  CHECK(hipMalloc(&A, N * sizeof(int)));
  CHECK(hipMalloc(&zwischenergebnisse, nblocks * sizeof(int)));
  CHECK(hipMalloc(&ergebnis, sizeof(int)));
  fill<<<nblocks, NTHREADS>>>(A, 1, N);
  hipDeviceSynchronize();
  sum<<<nblocks, NTHREADS>>>(A, zwischenergebnisse);
  hipDeviceSynchronize();
  assert(nblocks <= NTHREADS);
  sum<<<1, nblocks>>>(zwischenergebnisse, ergebnis);
  hipDeviceSynchronize();
  result_cpu = (int *)malloc(sizeof(int));
  if (result_cpu == NULL) {
    printf("allocation failed");
    exit(1);
  }

  CHECK(hipMemcpy(result_cpu, ergebnis, sizeof(int), hipMemcpyDeviceToHost));
  printf("%d\n", *result_cpu);
}
